/**
 * init.cu
 *
 * Created by Dimitrios Karageorgiou,
 *  for course "Parallel And Distributed Systems".
 *  Electrical and Computers Engineering Department, AuTh, GR - 2017-2018
 *
 * Code that will trigger loading of CUDA libraries.
 *
 * Version: 0.1
 */


#include <hip/hip_runtime.h>
#include <cassert>


namespace cuda
{

__global__ void
initKernel(float *devTable)
{
    float lovelyAcc = 0;
    for (int i = 0; i < 4; ++i) lovelyAcc += *(devTable + sizeof(float) * i);
    lovelyAcc = pow(lovelyAcc, 2.0);
}

void
deviceInit()
{
    float someData[] = { 1.0, 3.0, 5.0, 10.0 };

    float *devTable;

    hipError_t cudaStat = hipMalloc(&devTable, sizeof(float) * 4);
    assert(hipSuccess == cudaStat);
    cudaStat = hipMemcpy(devTable, someData, 4, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat);

    initKernel<<<8, 32>>>(devTable);

    cudaStat = hipFree(devTable);
    assert(hipSuccess == cudaStat);
}

}
