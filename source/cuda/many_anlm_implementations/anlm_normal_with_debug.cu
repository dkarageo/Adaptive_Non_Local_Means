#include "hip/hip_runtime.h"
/**
 * anlm_normal.cu
 *
 * Created by Dimitrios Karageorgiou,
 *  for course "Parallel And Distributed Systems".
 *  Electrical and Computers Engineering Department, AuTh, GR - 2017-2018
 *
 * A CUDA implementation of Adaptive Non-Local Means(ANLM) algorithm that
 * trades-off memory for computation time. It is suitable for small and
 * moderate sized images.
 *
 * The GPU is expected to be able to hold at least patchH*patchW+8 copies of
 * the image.
 *
 * Version: 0.1
 */

#include <iostream>
#include <cmath>
#include <cassert>
#include <vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <hip/hip_runtime_api.h>
#include "DMat.hpp"


#define BLOCK_SIZE 256
// #define MIN_BLOCKS_PER_SM 6

namespace cuda
{

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void
gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


template <class T>
std::vector<T>
calculateGaussianFilter(int m, int n, T sigma);

std::vector<int>
divideInRegions(int *ids, int imgH, int imgW, std::vector<int> &regionSizes);

template <class T>
void
fillRegionalMatrices(const DMatExpanded<T> &dSrc,
                     const DMat<T> &dFiltersSigma,
                     const DMat<int> &dIds,
                     std::vector<int> &regionalIndices,
                     std::vector<int> &regionSizes,
                     int patchH,
                     int patchW,
                     std::vector<DMat<T>> &rSrc,
                     std::vector<DMat<T>> &rPatches,
                     std::vector<DMat<T>> &rFiltersSigma,
                     std::vector<DMat<DMatPos>> &origPos);

template<class T>
__global__ void
cudaAnlmKernel(DMat<T> dRSrc, DMat<T> dRPatches, DMat<T> dDst,
               DMat<DMatPos> dOriginalPos, DMat<T> dRFiltersSigma,
               int patchH, int patchW);

template <class T>
__global__ void
cudaDivideRegions(DMatExpanded<T> dSrc,
                  DMat<T> dFiltersSigma,
                  DMat<int> dIds,
                  DMat<int> dRegionalIndices,
                  int patchH,
                  int patchW,
                  DMat<T*> dRSrc,
                  DMat<T*> dRPatches,
                  DMat<int> dRPatchesPitch,
                  DMat<T*> dRFiltersSigma,
                  DMat<DMatPos*> dOrigPos);

template <class T>
__global__ void
cudaApplyBlur(DMat<T> dRSrc, DMat<T> dPatchBlur);

template <class T> __device__ inline T __anlm_exp(T x) { return (T) exp(x); }
template <> __device__ inline float __anlm_exp<float>(float x) { return expf(x); }
template <class T> __device__ inline T __anlm_pow(T x, T y) { return (T) pow(x, y); }
template <> __device__ inline float __anlm_pow(float x, float y) { return powf(x, y); }

// template <class T>
// __global__ void
// printMatrices(DMat<T> patches, DMat<T> src, DMat<T> sigmas, DMat<DMatPos> pos)
// {
//     printf("===============Patches Matrix=============\n");
//     for (int i = 0; i < patches.width; ++i) {
//         for (int j = 0; j < patches.height; ++j) {
//             printf("%f ", patches(j, i));
//         }
//         printf("\n");
//     }
//
//     printf("===============Src Matrix=============\n");
//     for (int i = 0; i < src.width; ++i) {
//         printf("%f ", src(0, i));
//     }
//     printf("\n");
//
//     printf("===============Sigmas Matrix=============\n");
//     for (int i = 0; i < sigmas.width; ++i) {
//         printf("%f ", sigmas(0, i));
//     }
//     printf("\n");
//
//     printf("===============Pos Matrix=============\n");
//     for (int i = 0; i < sigmas.width; ++i) {
//         printf("(%d %d)", pos(0, i).y, pos(0, i).x);
//     }
//     printf("\n");
// }

template <class T>
void
adaptiveNonLocalMeansNormalData(T *src, T *dst, int *ids, T *filterSigma,
                                int imgH, int imgW, int patchH, int patchW,
                                T patchSigma, int regions)
{
    using std::vector;

    hipProfilerStart();

    // std::cout << "Entering anlm" << std::endl;
    // std::cout << "imgH:" << imgH << " imgW: " << imgW << " patchH:" << patchH
    //           << " patchW:" << patchW << " patchSigma:" << patchSigma
    //           << " regions:" << regions << std::endl;

    // Create generic matrices on device.
    DMatExpanded<T> dSrc(src, imgW, imgH, patchW, patchH);
    DMat<T>         dDst(imgW, imgH);
    DMat<int>       dIds(ids, imgW, imgH);
    DMat<T>         dFilterSigma(filterSigma, imgW, imgH);
    DMat<T>         dPatchBlur(patchW, patchH);

    // std::cout << "Created generic matrices" << std::endl;

    // DEBUG - Print src matrix
    // std::cout << "Src matrix" << std::endl;
    // for (int i = 0; i < dSrc.height; ++i) {
    //     for (int j = 0; j < dSrc.width; ++j) {
    //         std::cout << src[i*dSrc.width+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // DEBUG - Print ids matrix
    // std::cout << "Ids matrix" << std::endl;
    // for (int i = 0; i < dSrc.height; ++i) {
    //     for (int j = 0; j < dSrc.width; ++j) {
    //         std::cout << ids[i*dSrc.width+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // Find the index of each pixel in its own region and also get the size
    // of each region.
    vector<int> regionSizes(regions);
    vector<int> regionalIndices = divideInRegions(ids, imgH, imgW, regionSizes);

    // DEBUG - Print region sizes.
    // std::cout << "regionSizes" << std::endl;
    // for (int i = 0; i < regionSizes.size(); ++i) {
    //     std::cout << regionSizes[i] << " ";
    // }
    // std::cout << std::endl;
    // DEBUG - Print regionalIndices
    // std::cout << "regionalIndices" << std::endl;
    // for (int i = 0; i < dSrc.height; ++i) {
    //     for (int j = 0; j < dSrc.width; ++j) {
    //         std::cout << regionalIndices[i*dSrc.width+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // std::cout << "Found regions of each pixel" << std::endl;

    // Create separate matrices for each region on device.
    vector<DMat<T>>       dRegionsPatches(regions);
    vector<DMat<T>>       dRegionsSrc(regions);
    vector<DMat<T>>       dRegionsFiltersSigma(regions);
    vector<DMat<DMatPos>> dOriginalPos(regions);
    for (int i = 0; i < regions; ++i) {
        // Pixels of each region horizontally, pixels of the patch vertically.
        dRegionsPatches[i]      = DMat<T>(regionSizes[i], patchH*patchW);
        dRegionsSrc[i]          = DMat<T>(regionSizes[i], 1);
        dRegionsFiltersSigma[i] = DMat<T>(regionSizes[i], 1);
        dOriginalPos[i]         = DMat<DMatPos>(regionSizes[i], 1);
    }

    // std::cout << "Created special matrices" << std::endl;

    fillRegionalMatrices<T>(dSrc, dFilterSigma, dIds, regionalIndices, regionSizes,
                            patchH, patchW, dRegionsSrc, dRegionsPatches,
                            dRegionsFiltersSigma, dOriginalPos
    );

    // std::cout << "Matrices divided and copied" << std::endl;

    // Create a separate stream for each region.
    vector<hipStream_t> streams(regions);
    for (int i = 0; i < streams.size(); ++i) {
        hipError_t cudaStatus = hipStreamCreate(&streams[i]);
        assert(hipSuccess == cudaStatus);
    }

    // std::cout << "Streams created" << std::endl;

    // Calculate block and grid dims for each region.
    std::vector<dim3> rBlockDims(regions);
    std::vector<dim3> rGridDims(regions);
    for (int i = 0; i < regions; ++i) {
        int gridW = regionSizes[i] / BLOCK_SIZE;
        if ((regionSizes[i] % BLOCK_SIZE) > 0) ++gridW;
        rBlockDims[i] = dim3(BLOCK_SIZE);
        rGridDims[i]  = dim3(gridW);
    }

    // std::cout << "Block size calculated" << std::endl;

    // Calculate and apply a gaussian filter.
    std::vector<T> patchBlur = calculateGaussianFilter<T>(
            patchH, patchW, patchSigma);
    dPatchBlur.copyFromHost(patchBlur.data(), patchW, patchH);
    for (int i = 0; i < regions; ++i) {
        cudaApplyBlur<<<rGridDims[i], rBlockDims[i], 0, streams[i]>>>(
            dRegionsPatches[i], dPatchBlur);
    }

    // std::cout << "Applied gauss blur" << std::endl;

    // for (int i = 0; i < regions; ++i) {
    //     printMatrices<<<1, 1>>>(dRegionsPatches[i], dRegionsSrc[i], dRegionsFiltersSigma[i], dOriginalPos[i]);
    // }

    hipError_t cudaStat = hipFuncSetCacheConfig(reinterpret_cast<const void*>(
            cudaAnlmKernel<T>), hipFuncCachePreferL1);
    assert(hipSuccess == cudaStat);

    // Apply anlm to each region separately.
    for (int i = 0; i < regions; ++i) {
        cudaAnlmKernel<<<rGridDims[i], rBlockDims[i], 0, streams[i]>>>(
                dRegionsSrc[i], dRegionsPatches[i], dDst, dOriginalPos[i],
                dRegionsFiltersSigma[i], patchH, patchW
        );
        // gpuErrchk( hipPeekAtLastError() );
    }

    // std::cout << "Computed anlm" << std::endl;

    dDst.copyToHost(dst);

    for (int i = 0; i < streams.size(); ++i) hipStreamDestroy(streams[i]);

    // DEBUG - Print dst matrix
    // std::cout << "Dst matrix" << std::endl;
    // for (int i = 0; i < dSrc.height; ++i) {
    //     for (int j = 0; j < dSrc.width; ++j) {
    //         std::cout << dst[i*dSrc.width+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    // std::cout << "Returning anlm" << std::endl;

    hipProfilerStop();
}

template<class T>
__global__ void
// __launch_bounds__(BLOCK_SIZE, MIN_BLOCKS_PER_SM)
cudaAnlmKernel(DMat<T> dRSrc, DMat<T> dRPatches, DMat<T> dDst,
               DMat<DMatPos> dOriginalPos, DMat<T> dRFiltersSigma,
               int patchH, int patchW)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    const int patchSize  = patchH * patchW;
    const int regionSize = dRPatches.width;
    const T fSigma       = dRFiltersSigma(0, x) * dRFiltersSigma(0, x);

    T nom = 0;
    T denom = 0;
    T maxWeight = 0;

    __shared__ T sPatches[BLOCK_SIZE];
    const int patchesC = BLOCK_SIZE / patchSize;   // Patches that fit into shared memory.
    const int chunks   = regionSize / patchesC;    // Iterations needed to load all patches (without division remnants).
    const int targetY  = threadIdx.x % patchSize;  // Pixel in patch that current thread will load.
    const int targetX  = threadIdx.x / patchSize;  // Patch whose pixel current thread will load.

    // Calculate weights excluding the remainder patches (of chunks division).
    for (int cI = 0; cI < chunks; ++cI) {
        __syncthreads();
        sPatches[threadIdx.x] = dRPatches(targetY, cI*patchesC+targetX);
        __syncthreads();
        if (x >= regionSize) continue;

        // Iterate over all patches in shared memory.
        for (int i = 0; i < patchesC; ++i) {
            if (patchesC*cI+i == x) continue;  // Do not count the weight against itself.

            T weight = 0;
            for (int j = 0; j < patchSize; ++j) {
                T d = dRPatches(j, x) - sPatches[i*patchSize+j];
                weight += d*d;
            }
            weight = __anlm_exp<T>(-weight/fSigma);
            if (weight > maxWeight) maxWeight = weight;  // will use max weight for itself

            nom   += dRSrc(0, patchesC*cI+i) * weight;
            denom += weight;
        }
    }

    const int rem = regionSize % patchesC;  // Remainder of patches.

    // Load the remainder patches (if any).
    __syncthreads();
    if (threadIdx.x < rem*patchSize) {
        sPatches[threadIdx.x] = dRPatches(targetY, chunks*patchesC+targetX);
    }
    __syncthreads();
    if (x >= regionSize) return;  // Shared memory loads are over - thread no longer needed.

    // Calculate weights for the remainder patches.
    for (int i = 0; i < rem; ++i) {
        if (chunks*patchesC+i == x) continue;  // Do not count the weight against itself.

        T weight = 0;
        for (int j = 0; j < patchSize; ++j) {
            T d = dRPatches(j, x) - sPatches[i*patchSize+j];
            weight += d*d;
        }
        weight = __anlm_exp<T>(-weight/fSigma);
        if (weight > maxWeight) maxWeight = weight;  // will use max weight for itself

        nom   += dRSrc(0, patchesC*chunks+i) * weight;
        denom += weight;
    }

    // Guarantee that maxWeight will not be 0.
    if (maxWeight < __anlm_pow<T>(2.0, -52.0)) maxWeight = __anlm_pow<T>(2.0, -52.0);

    // Use maxWeight as the weight for itself.
    nom   += dRSrc(0, x) * maxWeight;
    denom += maxWeight;

    // Write back to the complete matrix.
    if (denom != 0) dDst.at(dOriginalPos(0, x)) = nom / denom;
    else dDst.at(dOriginalPos(0, x)) = dRSrc(0, x);
}

// template<class T>
// __global__ void
// // __launch_bounds__(BLOCK_SIZE)
// cudaAnlmKernel(DMat<T> dRSrc, DMat<T> dRPatches, DMat<T> dDst,
//                DMat<DMatPos> dOriginalPos, DMat<T> dRFiltersSigma,
//                int patchH, int patchW)
// {
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//
//     T fSigma = dRFiltersSigma(0, x);
//     fSigma = fSigma * fSigma;
//
//     T nom = 0;
//     T denom = 0;
//     T maxWeight = 0;
//
//     __shared__ T sPatches[BLOCK_SIZE];
//     int patchesC = BLOCK_SIZE / dRPatches.height;   // Patches that fit into shared memory.
//     int chunks   = dRPatches.width / patchesC;      // Iterations needed to load all patches (without division remnants).
//     int targetY  = threadIdx.x % dRPatches.height;  // Pixel in patch that current thread will load.
//     int targetX  = threadIdx.x / dRPatches.height;  // Patch whose pixel current thread will load.
//
//     // Calculate weights excluding the remainder patches of
//     // dRPatches.width/patchesC division.
//     for (int cI = 0; cI < chunks; ++cI) {
//         __syncthreads();
//         sPatches[threadIdx.x] = dRPatches(targetY, cI*patchesC+targetX);
//         __syncthreads();
//         if (x >= dRSrc.width) continue;
//
//         // Iterate over all patches in shared memory.
//         for (int i = 0; i < patchesC; ++i) {
//             if (patchesC*cI+i == x) continue;  // Do not count the weight against itself.
//
//             T weight = 0;
//             for (int j = 0; j < dRPatches.height; ++j) {
//                 T d = dRPatches(j, x) - sPatches[i*dRPatches.height+j];
//                 weight += d*d;
//             }
//             weight = __anlm_exp<T>(-weight/fSigma);
//             if (weight > maxWeight) maxWeight = weight;  // will use max weight for itself
//
//             nom += dRSrc(0, patchesC*cI+i) * weight;
//             denom += weight;
//         }
//     }
//
//     // Load the remainder patches (if any).
//     int rem = dRPatches.width % patchesC;
//     __syncthreads();
//     if (threadIdx.x < rem*dRPatches.height) {
//         sPatches[threadIdx.x] = dRPatches(targetY, chunks*patchesC+targetX);
//     }
//     __syncthreads();
//     if (x >= dRSrc.width) return;
//
//     // Calculate weights for the remainder patches of
//     // dRPatches.width/patchesC division.
//     for (int i = 0; i < rem; ++i) {
//         if (chunks*patchesC+i == x) continue;  // Do not count the weight against itself.
//
//         T weight = 0;
//         for (int j = 0; j < dRPatches.height; ++j) {
//             T d = dRPatches(j, x) - sPatches[i*dRPatches.height+j];
//             weight += d*d;
//         }
//         weight = __anlm_exp<T>(-weight/fSigma);
//         if (weight > maxWeight) maxWeight = weight;  // will use max weight for itself
//
//         nom += dRSrc(0, patchesC*chunks+i) * weight;
//         denom += weight;
//     }
//
//     // Guarantee that maxWeight will not be 0.
//     if (maxWeight < __anlm_pow<T>(2.0, -52.0)) maxWeight = __anlm_pow<T>(2.0, -52.0);
//
//     // Use maxWeight as the weight for itself.
//     nom += dRSrc(0, x) * maxWeight;
//     denom += maxWeight;
//
//     // Write back to the complete matrix.
//     if (denom != 0) dDst.at(dOriginalPos(0, x)) = nom / denom;
//     else dDst.at(dOriginalPos(0, x)) = dRSrc(0, x);
// }

// template<class T>
// __global__ void
// //__launch_bounds__(BLOCK_SIZE, MIN_BLOCKS_PER_SM)
// cudaAnlmKernel(DMat<T> dRSrc, DMat<T> dRPatches, DMat<T> dDst,
//                DMat<DMatPos> dOriginalPos, DMat<T> dRFiltersSigma,
//                int patchH, int patchW)
// {
//     int x = blockIdx.x * blockDim.x + threadIdx.x;
//     if (x >= dRSrc.width) return;
//
//     T fSigma = dRFiltersSigma(0, x);
//     fSigma = fSigma * fSigma;
//
//     T nom = 0;
//     T denom = 0;
//     T maxWeight = 0;
//
//     // Iterate over all pixels in the region to
//     for (int i = 0; i < dRPatches.width; ++i) {
//         if (i == x) continue;  // Do not count the weight against itself.
//
//         T weight = 0;
//         for (int j = 0; j < dRPatches.height; ++j) {
//             T d = dRPatches(j, x) - dRPatches(j, i);
//             weight += d*d;
//         }
//         weight = __anlm_exp<T>(-weight/fSigma);
//         if (weight > maxWeight) maxWeight = weight;  // will use max weight for itself
//
//         nom += dRSrc(0, i) * weight;
//         denom += weight;
//     }
//
//     // Guarantee that maxWeight will not be 0.
//     if (maxWeight < __anlm_pow<T>(2.0, -52.0)) maxWeight = __anlm_pow<T>(2.0, -52.0);
//
//     // Use maxWeight as the weight for itself.
//     nom += dRSrc(0, x) * maxWeight;
//     denom += maxWeight;
//
//     // Write back to the complete matrix.
//     if (denom != 0) dDst.at(dOriginalPos(0, x)) = nom / denom;
//     else dDst.at(dOriginalPos(0, x)) = dRSrc(0, x);
// }


/**
 * Kernel that applies given gaussian blur to patches contained in dRSrc.
 */
template <class T>
__global__ void
cudaApplyBlur(DMat<T> dRSrc, DMat<T> dPatchBlur)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= dRSrc.width) return;

    const int patchW = dPatchBlur.width;

    for (int i = 0; i < dPatchBlur.height; ++i) {
        for (int j = 0; j < patchW; ++j) {
            dRSrc(i*patchW+j, x) *= dPatchBlur(i, j);
        }
    }
}

/**
 * Kernel that copies data from given uniform matrices into given region
 * specific matrices.
 */
template <class T>
__global__ void
cudaDivideRegions(DMatExpanded<T> dSrc,
                  DMat<T> dFiltersSigma,
                  DMat<int> dIds,
                  DMat<int> dRegionalIndices,
                  int patchH,
                  int patchW,
                  DMat<T*> dRSrc,
                  DMat<T*> dRPatches,
                  DMat<int> dRPatchesPitch,
                  DMat<T*> dRFiltersSigma,
                  DMat<DMatPos*> dOrigPos)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dSrc.width || y >= dSrc.height) return;

    const int currentId = dIds(y, x);  // id of region
    const int rIndex    = dRegionalIndices(y, x);  // location in region

    const int boundH = (patchH - 1) / 2;
    const int boundW = (patchW - 1) / 2;

    // Keep a reference of pixel's original position in dSrc.
    *(dOrigPos(0, currentId) + rIndex) = DMatPos(y, x);

    // Copy filter sigma.
    *(dRFiltersSigma(0, currentId) + rIndex) = dFiltersSigma(y, x);

    // Copy the current pixel.
    *(dRSrc(0, currentId) + rIndex) = dSrc(y, x);

    const int patchesPitch = dRPatchesPitch(0, currentId);

    // Copy the entire neighborhood of current pixel.
    for (int i = -boundH; i < boundH+1; ++i) {
        for (int j = -boundW; j < boundW+1; ++j) {
            int row = (i+boundH)*patchW + (j+boundW);
            T *loc = (T*) ((char*) dRPatches(0, currentId) + row*patchesPitch) + rIndex;
            *loc = dSrc(y+i, x+j);
        }
    }
}

/**
 * Returns a vector, that contains for each element in ids the index of that
 * pixel in its own region.
 *
 * The region of a pixel is consisted of the pixels having the same id.
 *
 * Parameters:
 *  -ids : Pointer to an array of size imgH*imgW containing the id of each pixel
 *          in row major order.
 *  -imgH : Height of the image in pixels (number of rows in ids).
 *  -imgW : Width of the image in pixels (number of columns in ids).
 *  -regionSizes : A vector of size equal to the number of different regions
 *          (i.e. different ID values) contained in ids. On return, each
 *          item in this array will contain the number of elements in the
 *          corresponding region.
 *
 * Returns:
 *  A vector of size imgH*imgW containing the region-wise indices for each pixel.
 *  Also, in regionSizes returns the number of elements in each region.
 */
std::vector<int>
divideInRegions(int *ids, int imgH, int imgW, std::vector<int> &regionSizes)
{
    std::vector<int> indices(imgH*imgW);

    for (int i = 0; i < regionSizes.size(); ++i) regionSizes[i] = 0;

    for (int i = 0; i < imgH; ++i) {
        for (int j = 0; j < imgW; ++j) {
            int index = imgW * i + j;
            int id = *(ids+index);
            indices[index] = regionSizes[id];
            ++regionSizes[id];
        }
    }

    return indices;
}

/**
 * Divides given uniform matrices into region specific matrices.
 */
template <class T>
void
fillRegionalMatrices(const DMatExpanded<T> &dSrc,
                     const DMat<T> &dFiltersSigma,
                     const DMat<int> &dIds,
                     std::vector<int> &regionalIndices,
                     std::vector<int> &regionSizes,
                     int patchH,
                     int patchW,
                     std::vector<DMat<T>> &rSrc,
                     std::vector<DMat<T>> &rPatches,
                     std::vector<DMat<T>> &rFiltersSigma,
                     std::vector<DMat<DMatPos>> &origPos)
{
    using std::vector;

    // Objects cannot be copied directly to device, so the only way to pass
    // an array of objects is to pass their raw device pointers.
    vector<T*>       rawSrc(rSrc.size());
    vector<T*>       rawPatches(rPatches.size());
    vector<int>      rawPatchesPitch(rPatches.size());
    vector<T*>       rawFiltersSigma(rFiltersSigma.size());
    vector<DMatPos*> rawOrigPos(origPos.size());

    for (int i = 0; i < rSrc.size(); ++i) {
        rawSrc[i]          = rSrc[i].data;
        rawPatches[i]      = rPatches[i].data;
        rawPatchesPitch[i] = rPatches[i].pitch;
        rawFiltersSigma[i] = rFiltersSigma[i].data;
        rawOrigPos[i]      = origPos[i].data;
    }

    // Wrap the pointers into DMats, so they can be easily passed to device.
    DMat<T*>       dRPatches(rawPatches.data(), rawPatches.size(), 1);
    DMat<int>      dRPatchesPitch(rawPatchesPitch.data(), rawPatches.size(), 1);
    DMat<T*>       dRSrc(rawSrc.data(), rawSrc.size(), 1);
    DMat<T*>       dRFiltersSigma(rawFiltersSigma.data(), rawFiltersSigma.size(), 1);
    DMat<DMatPos*> dOrigPos(rawOrigPos.data(), rawOrigPos.size(), 1);

    DMat<int> dRegionalIndices(regionalIndices.data(), dSrc.width, dSrc.height);

    int gridW = dSrc.width / 32;
    if ((dSrc.width % 32) > 0) gridW++;
    int gridH = dSrc.height / (BLOCK_SIZE / 32);
    if ((dSrc.height % (BLOCK_SIZE / 32)) > 0) gridH++;
    dim3 blockDim(32, BLOCK_SIZE / 32);
    dim3 gridDim(gridW, gridH);

    cudaDivideRegions<<<gridDim, blockDim>>>(
        dSrc, dFiltersSigma, dIds, dRegionalIndices, patchH, patchW,
        dRSrc, dRPatches, dRPatchesPitch, dRFiltersSigma, dOrigPos
    );
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();
}

/**
 * Calculates a gaussian filter matrix.
 *
 * Parameters:
 *  -m : Height of filter matrix.
 *  -n : Width of filter matrix.
 *  -sigma : Sigma to be used for gaussian filter computation.
 *
 * Returns:
 *  A vector containing the gaussian filter in row major order.
 */
template <class T>
std::vector<T>
calculateGaussianFilter(int m, int n, T sigma)
{
    std::vector<T> filter(m*n);
    T sum = 0;
    T mean = (m - 1) / 2;

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            T a = (T) i - mean;
            T b = (T) j - mean;
            T val = exp(-(a*a + b*b) / (2*sigma*sigma));
            filter[i*m+j] = val;
            sum += val;
        }
    }

    // Normalize kernels.
    T max = filter[m*n/2] / sum;  // greatest kernel will always be on center
    for (int i = 0; i < m*n; ++i) filter[i] /= (sum * max);

    return filter;
}


// ----------- Declarations for pregenerating code by compiler -------------
template
void
adaptiveNonLocalMeansNormalData<float>(float *, float *, int *, float *,
                                       int, int, int, int, float, int);
template
void
adaptiveNonLocalMeansNormalData<double>(double *, double *, int *, double *,
                                        int, int, int, int, double, int);

}
