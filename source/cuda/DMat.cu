/**
 * DMat.cu
 *
 * Version: 0.1
 */

#include <iostream>
#include <assert.h>
#include "DMat.hpp"


namespace cuda {


template <class T>
void
swap(DMat<T>& first, DMat<T>& second)
{
    using std::swap;

    swap(first.width, second.width);
    swap(first.height, second.height);
    swap(first.pitch, second.pitch);
    swap(first.data, second.data);
    swap(first._isOwner, second._isOwner);
}

template <class T>
DMat<T>::DMat()
: _isOwner(true),
  height(0),
  width(0),
  pitch(0),
  data(nullptr)
{
    // std::cout << "Calling DMat constructor" << std::endl;
}

template <class T>
DMat<T>::DMat(size_t width, size_t height)
: DMat()
{
    hipError_t cudaStat;

    this->width = width;
    this->height = height;

    if (width > 0 && height > 0) {
        // std::cout << "Creating matrix: " << width << "x" << height << std::endl;
        cudaStat = hipMallocPitch(&data, &pitch, sizeof(T) * width, height);
        assert(hipSuccess == cudaStat);
    } else std::cout << "Creating empty matrix" << std::endl;
}

template <class T>
DMat<T>::DMat(T* hostData, size_t width, size_t height)
: DMat(width, height)
{
    copyFromHost(hostData, width, height);
}

template <class T>
DMat<T>::DMat(const DMat<T> &dmat)
{
    // std::cout << "Copying matrix - isOwner:" << (dmat._isOwner ? "true" : "false")
    //           << " width:" << dmat.width
    //           << " height:" << dmat.height
    //           << " pitch:" << dmat.pitch
    //           << std::endl;

    data = dmat.data;
    width = dmat.width;
    height = dmat.height;
    pitch = dmat.pitch;
    _isOwner = false;
}

template <class T>
DMat<T>::DMat(DMat<T> &&dmat)
{
    swap<T>(*this, dmat);
}

template <class T>
DMat<T>&
DMat<T>::operator=(DMat<T> dmat)
{
    swap<T>(*this, dmat);
    if (data == dmat.data) dmat._isOwner = false;
    return *this;
}

template <class T>
DMat<T>::~DMat()
{
    // std::cout << "Destructing DMat" << std::endl;

    if (_isOwner && data) {
        // std::cout << "is owner" << std::endl;

        hipError_t cudaStat = hipFree(data);
        assert(hipSuccess == cudaStat);
    }
}

template <class T>
void
DMat<T>::copyFromHost(T *hostData, int width, int height)
{
    hipError_t cudaStat;

    cudaStat = hipMemcpy2D(data, pitch,
                            hostData, width*sizeof(T),
                            width*sizeof(T), height,
                            hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat);
}

template <class T>
void
DMat<T>::copyToHost(T *hostDst)
{
    hipError_t cudaStat;

    cudaStat = hipMemcpy2D(hostDst, width*sizeof(T),
                            data, pitch,
                            width*sizeof(T), height,
                            hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat);
}

template <class T>
DMatExpanded<T>::DMatExpanded(size_t width, size_t height,
                              size_t expWidth, size_t expHeight)
{
    this->width = width;
    this->height = height;
    _expWidth = width + expWidth * 2;
    _expHeight = height + expHeight * 2;

    hipError_t cudaStat;

    // Allocate a table large enough for the expanded matrix.
    cudaStat = hipMallocPitch(&_expData, &pitch,
                               sizeof(T) * _expWidth, _expHeight);
    assert(hipSuccess == cudaStat);

    // Point DMat data to the portion of original table, so DMatExpanded
    // can be a valid DMat too. A displacement of _expHeight rows, followed
    // by _expWidth columns, will do the job. Expansion cells will be seen as
    // pitch padding by DMat.
    data = (T*) ((char*) _expData + expHeight*pitch) + expWidth;
}

template <class T>
DMatExpanded<T>::DMatExpanded(T *hostData, size_t width, size_t height,
                              size_t expWidth, size_t expHeight)
: DMatExpanded(width, height, expWidth, expHeight)
{
    copyFromHost(hostData, width, height, expWidth, expHeight);
}

template <class T>
DMatExpanded<T>::DMatExpanded(const DMatExpanded<T> &dmat)
: DMat<T>(dmat)
{
    _expData = dmat._expData;
    _expWidth = dmat._expWidth;
    _expHeight = dmat._expHeight;
}

template <class T>
DMatExpanded<T>&
DMatExpanded<T>::operator=(const DMatExpanded<T> &dmat)
{
    DMat<T>::operator=(dmat);
    _expData = dmat._expData;
    _expWidth = dmat._expWidth;
    _expHeight = dmat._expHeight;
    return *this;
}

template <class T>
DMatExpanded<T>::~DMatExpanded()
{
    // Restore data attr for parent destructor.
    data = _expData;
}

template <class T>
void
DMatExpanded<T>::copyFromHost(T *hostData, int width, int height,
                              int vExpand, int hExpand)
{
    hipError_t cudaStat;

    // Copy original table to the centre of expanded matrix.
    DMat<T>::copyFromHost(hostData, width, height);

    // Expand horizontally.
    for (int i = 0; i < hExpand; ++i) {
        cudaStat = hipMemcpy2D(data-i-1, pitch, data+i, pitch, sizeof(T),
                                height, hipMemcpyDeviceToDevice);
        assert(hipSuccess == cudaStat);
        cudaStat = hipMemcpy2D(data+width+i, pitch, data+width-i-1, pitch, sizeof(T),
                                height, hipMemcpyDeviceToDevice);
        assert(hipSuccess == cudaStat);
    }

    // Expand vertically.
    for (int i = 0; i < vExpand; ++i) {
        T *dst = (T*) ((char*) (data - hExpand) - pitch * (i + 1));
        T *src = (T*) ((char*) (data - hExpand) + pitch * i);
        cudaStat = hipMemcpy2D(dst, pitch, src, pitch,
                                (width+hExpand*2)*sizeof(T),
                                1, hipMemcpyDeviceToDevice);
        assert(hipSuccess == cudaStat);
        dst = (T*) ((char*) (data - hExpand) + pitch * (height + i));
        src = (T*) ((char*) (data - hExpand) + pitch * (height - i - 1));
        cudaStat = hipMemcpy2D(dst, pitch, src, pitch,
                                (width+hExpand*2)*sizeof(T),
                                1, hipMemcpyDeviceToDevice);
        assert(hipSuccess == cudaStat);
    }
}


template class DMat<double>;
template class DMat<double*>;
template class DMat<int>;
template class DMat<int*>;
template class DMat<float>;
template class DMat<float*>;
template class DMat<DMatPos>;
template class DMat<DMatPos*>;
template class DMat<DMat<double>>;
template class DMat<DMat<float>>;
template class DMat<DMat<DMatPos>>;

template class DMatExpanded<double>;
template class DMatExpanded<double*>;
template class DMatExpanded<int>;
template class DMatExpanded<int*>;
template class DMatExpanded<float>;
template class DMatExpanded<float*>;
template class DMatExpanded<DMatPos>;
template class DMatExpanded<DMatPos*>;


}
