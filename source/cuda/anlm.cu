#include "hip/hip_runtime.h"
/**
 * anlm.cu
 *
 * Version: 0.1
 */

#include <iostream>
#include <cmath>
#include <vector>
#include "DMat.hpp"

#define BLOCK_SIZE 256


namespace cuda
{

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

template <class T>
__global__ void
cudaFindNextEqualKernel(DMat<int> dIds, DMat<DMatPos> dNext, DMat<DMatPos> dPrev);
template <class T>
__global__ void
cudaSetPrevEqualKernel(DMat<DMatPos> dNext, DMat<DMatPos> dPrev);
template <class T>
std::vector<T>
calculateGaussianFilter(int m, int n, T sigma);

template<class T>
__global__ void
cudaAnlmKernel(DMatExpanded<T> dSrc, DMat<T> dDst, DMat<DMatPos> dNext,
               DMat<DMatPos> dPrev, DMat<T> dFilterSigma, DMat<T> dPatchBlur,
               int patchH, int patchW);


template <class T>
void
adaptiveNonLocalMeans(T *src, T *dst, int *ids, T *filterSigma,
                     int imgH, int imgW, int patchH, int patchW,
                     T patchSigma)
{
    std::cout << "Entering anlm" << std::endl;
    std::cout << "imgH:" << imgH << " imgW: " << imgW << " patchH:" << patchH
              << " patchW:" << patchW << " patchSigma:" << patchSigma << std::endl;

    // Create matrices on device.
    DMatExpanded<T> dSrc(src, imgW, imgH, patchW, patchH);
    DMat<T> dDst(imgW, imgH);
    DMat<int> dIds(ids, imgW, imgH);
    DMat<DMatPos> dNext(imgW, imgH);
    DMat<DMatPos> dPrev(imgW, imgH);
    DMat<T> dFilterSigma(filterSigma, imgW, imgH);
    DMat<T> dPatchBlur(patchW, patchH);

    std::cout << "Matrices created" << std::endl;

    int gridW = imgW / 32;
    if ((imgW % 32) > 0) gridW++;
    int gridH = imgH / (BLOCK_SIZE / 32);
    if ((imgH % (BLOCK_SIZE / 32)) > 0) gridH++;
    dim3 blockDim(32, BLOCK_SIZE / 32);
    dim3 gridDim(gridW, gridH);

    std::cout << "Block size calculated" << std::endl;

    // Precompute the pixels belonging to each search area.
    cudaFindNextEqualKernel<T><<<gridDim, blockDim>>>(dIds, dNext, dPrev);
    cudaSetPrevEqualKernel<T><<<gridDim, blockDim>>>(dNext, dPrev);

    // While computing search areas on GPU, calculate a gaussian filter on CPU.
    std::vector<T> patchBlur = calculateGaussianFilter<T>(
            patchH, patchW, patchSigma
    );
    dPatchBlur.copyFromHost(patchBlur.data(), patchW, patchH);
    hipDeviceSynchronize();

    std::cout << "Computed nexts and gauss blur" << std::endl;

    // Apply anlm to each pixel separately.
    cudaAnlmKernel<<<gridDim, blockDim>>>(
            dSrc, dDst, dNext, dPrev, dFilterSigma, dPatchBlur, patchH, patchW
    );
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();

    std::cout << "Computed anlm" << std::endl;

    dDst.copyToHost(dst);

    std::cout << "Returning anlm" << std::endl;
}

template<class T>
__global__ void
cudaAnlmKernel(DMatExpanded<T> dSrc, DMat<T> dDst, DMat<DMatPos> dNext,
               DMat<DMatPos> dPrev, DMat<T> dFilterSigma, DMat<T> dPatchBlur,
               int patchH, int patchW)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dSrc.width || y >= dSrc.height) return;

    const int boundH = (patchH - 1) / 2;
    const int boundW = (patchW - 1) / 2;
    const T fSigma = dFilterSigma(y, x) * dFilterSigma(y, x);

    T nom = 0;
    T denom = 0;
    T maxWeight = 0;

    DMatPos curCell = dNext(y, x);
    bool forward = true;
    if (curCell.x == -1 || curCell.y == -1) {  // True only for last pixel of each region.
        curCell = dPrev(y, x);
        forward = false;
    }

    // if (x == 0 && y == 0) {
    //     for (int i = 0; i < dNext.height; ++i) {
    //         for (int j = 0; j < dNext.width; ++j) {
    //             DMatPos n = dNext(i, j);
    //             DMatPos p = dPrev(i, j);
    //             printf("%d,%d: %d,%d %d,%d\n", i, j, p.y, p.x, n.y, n.x);
    //         }
    //     }
    // }

    // if (x == 0 && y == 0) {
    //     for (int i = 0; i < dFilterSigma.width; ++i) {
    //         for (int j = 0; j < dFilterSigma.height; ++j) {
    //             printf("%f ", dFilterSigma(i, j));
    //         }
    //         printf("\n");
    //     }
    // }

    // int l = 0;

    // Iterate over all pixels in the region of (y, x) pixel.
    // for (int l = 0; l < 5000; ++l) {
    while (true) {
        // if (x == 0 && y == 0) {
        //     printf("%d cur: %d,%d\n", l, curCell.y, curCell.x);
        // }

        T weight = 0;

        for (int i = -boundH; i < boundH+1; ++i) {
            for (int j = -boundW; j < boundW+1; ++j) {
                // if (x == 0 && y == 0)
                //     printf("%d %d %f %f  ", i, j, dSrc(y+i, x+j)*dPatchBlur(boundH+i, boundW+j), dSrc(curCell.y+i, curCell.x+j)*dPatchBlur(boundH+i, boundW+j));

                T d = dSrc(y+i, x+j) * dPatchBlur(boundH+i, boundW+j) -
                      dSrc(curCell.y+i, curCell.x+j) * dPatchBlur(boundH+i, boundW+j);
                weight += d*d;

                // if (x== 0 && y == 0)
                //     printf("%f %f |", d*d, weight);
            }
            // if (x == 0 && y == 0) printf("\n");
        }

        // if (y == 0 && x == 0) {
        //     printf("%d: %.10f - %f\n", l, -weight/fSigma, fSigma);
        //     ++l;
        // }

        weight = exp(-weight/fSigma);


        nom += dSrc(curCell.y, curCell.x) * weight;
        denom += weight;
        if (weight > maxWeight) maxWeight = weight;

        curCell = forward ? dNext.at(curCell) : dPrev.at(curCell);

        // if (x == 10 && y == 10)
        //     printf("next: %d %d %s \n", curCell.y, curCell.x, forward ? "forw" : "back");

        // When no elements can be found forward, start going backwards.
        if (curCell.x == -1 || curCell.y == -1) {
            if (forward) {
                curCell = dPrev(y, x);  // Start going backwards from (y, x).
                forward = false;
                if (curCell.x == -1 || curCell.y == -1) break;
            } else {
                // printf("%d: %d %d %d %d\n", l, y, x, curCell.y, curCell.x);
                break;
            }
            // printf("%d: %d %d %d %d\n", l, y, x, curCell.y, curCell.x);
        }
    }

    if (maxWeight < pow(2.0, -52.0)) maxWeight = pow(2.0, -52.0);

    // if (y == 0 && x == 0)
    //     printf("Max weight: %.20f\n", maxWeight);

    // Calculate the weight with itself.
    nom += dSrc(y, x) * maxWeight;
    denom += maxWeight;

    // if (y == 0 && x == 0)
    //     printf("Orig: %.10f Result: %.10f - Denom: %f\n", dSrc(y, x), nom / denom, denom);

    if (denom != 0) dDst(y, x) = nom / denom;
    else dDst(y, x) == dSrc(y, x);
}

template <class T>
__global__ void
cudaFindNextEqualKernel(DMat<int> dIds, DMat<DMatPos> dNext, DMat<DMatPos> dPrev)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dIds.width || y >= dIds.height) return;

    T self = dIds(y, x);
    DMatPos nextPos(-1, -1);

    // Search the remnants of current row.
    for (int j = x+1; j < dIds.width; ++j) {
        if (dIds(y, j) == self) {
            nextPos.x = j;
            nextPos.y = y;
            goto eqFound;
        }
    }

    // Search all next rows.
    for (int i = y+1; i < dIds.height; ++i) {
        for (int j = 0; j < dIds.width; ++j) {
            if (dIds(i, j) == self) {
                nextPos.x = j;
                nextPos.y = i;
                goto eqFound;
            }
        }
    }
eqFound:
    dNext(y, x) = nextPos;
    // Initialize all previous positions.
    dPrev(y, x) = DMatPos(-1, -1);
}

template <class T>
__global__ void
cudaSetPrevEqualKernel(DMat<DMatPos> dNext, DMat<DMatPos> dPrev)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dNext.width || y >= dNext.height) return;
    DMatPos next = dNext(y, x);
    if (next.y != -1 && next.x != -1) dPrev.at(next) = DMatPos(y, x);
}

template <class T>
std::vector<T>
calculateGaussianFilter(int m, int n, T sigma)
{
    std::vector<T> filter(m*n);
    T sum = 0;
    T mean = (m - 1) / 2;

    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            T a = (T) i - mean;
            T b = (T) j - mean;
            T val = exp(-(a*a + b*b) / (2*sigma*sigma));
            filter[i*m+j] = val;
            sum += val;
        }
    }

    // Normalize kernels.
    T max = filter[m*n/2] / sum;  // greatest kernel will always be on center
    for (int i = 0; i < m*n; ++i) filter[i] /= (sum * max);

    return filter;
}


template
void
adaptiveNonLocalMeans<float> (float *, float *, int *, float *,
                              int, int, int, int, float);
template
void
adaptiveNonLocalMeans<double>(double *, double *, int *, double *,
                              int, int, int, int, double);

}
